
#include <hip/hip_runtime.h>
#include <cstdlib>

#define PI   3.1415926535897932384626433832795029f
#define PIx2 6.2831853071795864769252867665590058f

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define K_ELEMS_PER_GRID 2048

#define KERNEL_PHI_MAG_THREADS_PER_BLOCK 512
#define KERNEL_Q_THREADS_PER_BLOCK 256
#define KERNEL_Q_K_ELEMS_PER_GRID 1024


struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float PhiMag;
};

__constant__ __device__ kValues ck[KERNEL_Q_K_ELEMS_PER_GRID];

__global__ void
ComputePhiMag_GPU(float* phiR, float* phiI, float* phiMag, int numK) {
  int indexK = blockIdx.x*KERNEL_PHI_MAG_THREADS_PER_BLOCK + threadIdx.x;
  if (indexK < numK) {
    float real = phiR[indexK];
    float imag = phiI[indexK];
    phiMag[indexK] = real*real + imag*imag;
  }
}

__global__ void
ComputeQ_GPU(int numK, int kGlobalIndex,
	     float* x, float* y, float* z, float* Qr , float* Qi)
{
  __shared__ float sx,sy,sz,sQr,sQi;

  int xIndex = blockIdx.x*KERNEL_Q_THREADS_PER_BLOCK + threadIdx.x;

  sx = x[xIndex];
  sy = y[xIndex];
  sz = z[xIndex];
  sQr = Qr[xIndex];
  sQi = Qi[xIndex];

  int kIndex = 0;
  if (numK % 4)
 {
    for (int j=0;j<numK%4;j++)
   {
    float expArg = PIx2 * (ck[j].Kx * sx + ck[j].Ky * sy + ck[j].Kz * sz);
    sQr += ck[j].PhiMag * cos(expArg);
    sQi += ck[j].PhiMag * sin(expArg);
    kIndex++;
    kGlobalIndex++;
   }
 }

  for (; (kIndex < KERNEL_Q_K_ELEMS_PER_GRID) && (kGlobalIndex < numK);
       kIndex += 4, kGlobalIndex += 4) {
    float expArg = PIx2 * (ck[kIndex].Kx * sx +
			   ck[kIndex].Ky * sy +
			   ck[kIndex].Kz * sz);
    sQr += ck[kIndex].PhiMag * cos(expArg);
    sQi += ck[kIndex].PhiMag * sin(expArg);

    int kIndex1 = kIndex + 1;
    float expArg1 = PIx2 * (ck[kIndex1].Kx * sx +
			    ck[kIndex1].Ky * sy +
			    ck[kIndex1].Kz * sz);
    sQr += ck[kIndex1].PhiMag * cos(expArg1);
    sQi += ck[kIndex1].PhiMag * sin(expArg1);

    int kIndex2 =kIndex+ 2;
    float expArg2 = PIx2 * (ck[kIndex2].Kx * sx +
			    ck[kIndex2].Ky * sy +
			    ck[kIndex2].Kz * sz);
    sQr += ck[kIndex2].PhiMag * cos(expArg2);
    sQi += ck[kIndex2].PhiMag * sin(expArg2);   

    int kIndex3 =kIndex+ 3;
    float expArg3 = PIx2 * (ck[kIndex3].Kx * sx +
			    ck[kIndex3].Ky * sy +
			    ck[kIndex3].Kz * sz);
    sQr += ck[kIndex3].PhiMag * cos(expArg3);
    sQi += ck[kIndex3].PhiMag * sin(expArg3);   
  }

  Qr[xIndex] = sQr;
  Qi[xIndex] = sQi;
}

void computePhiMag_GPU(int numK, float* phiR_d, float* phiI_d, float* phiMag_d)
{
  int phiMagBlocks = (numK-1) / KERNEL_PHI_MAG_THREADS_PER_BLOCK+1;
  dim3 DimPhiMagBlock(KERNEL_PHI_MAG_THREADS_PER_BLOCK, 1);
  dim3 DimPhiMagGrid(phiMagBlocks, 1);

  ComputePhiMag_GPU <<< DimPhiMagGrid, DimPhiMagBlock >>> 
    (phiR_d, phiI_d, phiMag_d, numK);
}

void computeQ_GPU(int numK, int numX,
                  float* x_d, float* y_d, float* z_d,
                  kValues* kVals,
                  float* Qr_d, float* Qi_d)
{
  int QGrids = (numK-1)/KERNEL_Q_K_ELEMS_PER_GRID+1;
  int QBlocks =(numX-1)/KERNEL_Q_THREADS_PER_BLOCK+1;
  dim3 DimQBlock(KERNEL_Q_THREADS_PER_BLOCK, 1);
  dim3 DimQGrid(QBlocks, 1);

  for (int QGrid = 0; QGrid < QGrids; QGrid++) {
    int QGridBase = QGrid * KERNEL_Q_K_ELEMS_PER_GRID;
    kValues* kValsTile = kVals + QGridBase;
    int numElems = MIN(KERNEL_Q_K_ELEMS_PER_GRID, numK - QGridBase);
    hipMemcpyToSymbol(HIP_SYMBOL(ck), kValsTile, numElems * sizeof(kValues), 0);
    ComputeQ_GPU <<< DimQGrid, DimQBlock >>>
      (numK, QGridBase, x_d, y_d, z_d, Qr_d, Qi_d);
  }
}

void createDataStructsCPU(int numK, int numX, float** phiMag,
	 float** Qr, float** Qi)
{
  *phiMag = (float* ) malloc(numK * sizeof(float));
  *Qr = (float*) malloc(numX * sizeof (float));
  *Qi = (float*) malloc(numX * sizeof (float));
}
